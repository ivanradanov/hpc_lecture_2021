#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matmul.hpp"

__global__ void mat_mul(float *A, float *B, float *C, int size_x, int size_y)
{
	// these point to the first element of the first block we are considering
	float *a = A + blockIdx.y * BLOCK_SIZE * size_x;
	float *b = B + blockIdx.x * BLOCK_SIZE;
	float *c = C + blockIdx.y * BLOCK_SIZE * size_y + blockIdx.x * BLOCK_SIZE;


	int numBlocks = size_x / BLOCK_SIZE;

	float res;

	for (int i = 0; i < numBlocks; a += BLOCK_SIZE, b += BLOCK_SIZE * size_y, i++) {

		// now a and b point to the first element of the block we are considering

		__shared__ float sa[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float sb[BLOCK_SIZE][BLOCK_SIZE];

		sa[threadIdx.y][threadIdx.x] = a[size_x * threadIdx.y + threadIdx.x];
		sb[threadIdx.y][threadIdx.x] = b[size_y * threadIdx.y + threadIdx.x];

		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE; j++) {
			res += sa[threadIdx.y][j] * sb[j][threadIdx.x];
			//printf("%f\t%f\n", sa[threadIdx.y][j] , sb[j][threadIdx.x]);
		}
		__syncthreads();
	}
	//printf("%d, %d, %d, %d, %f\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, res);
	//printf("%f\n", res);
	c[threadIdx.x + threadIdx.y * size_y] = res;
}

void h_mat_mul(dim3 grid, dim3 block, float *A, float *B, float *C, int size_x, int size_y) {
	mat_mul<<< grid, block >>>(A, B, C, size_x, size_y);
}
